
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
using namespace std;

double f(double x, double y) {
    return x*(1-x) + y*(1-y);
}

double* allocate_array(int N, int M) {
    return (double*) malloc(sizeof(double)*N*M);
}

double* initialize_x(int N, double h) { // initialize solution, including boundary
    double* x = allocate_array(N+2, N+2);
    for (int i=0; i <= N+1; i++) {
        for (int j=0; j <= N+1; j++) {
            x[i*(N+2)+j] = 0;
        }
    }
    return x;
}

double* initialize_b(int N, double h) { // initialize right side
    double* b = allocate_array(N+2, N+2);
    double i_coord = 0;
    double j_coord = 0;
    for (int i=0; i <= N+1; i++) {
        for (int j=0; j <= N+1; j++) {
            b[i*(N+2)+j] = f(i_coord, j_coord);
            j_coord += h;
        }
        i_coord += h;
    }
    return b;
}

double largestDiff(double* x, double* y, int N) {
    double largestDiff = 0;
    for (int i=1; i <= N; i++) {
        for (int j=1; j <= N; j++) {
            double diff = abs(x[i*(N+2)+j] - y[i*(N+2)+j]) > largestDiff;
            if (diff > largestDiff) {
                largestDiff = diff;
            }
        }
    }
    return largestDiff;
}

double* jacobi(int N) {
    double h = 1.0 / (double) N;
    double* x_old = initialize_x(N, h);
    double* b = initialize_b(N, h);
    double* x = allocate_array(N+2, N+2);

    for (int iteration=0; iteration < 2000; iteration++) {
        // update of x
        for (int i=1; i <= N; i++) {
            for (int j=1; j <= N; j++) {
                x[i*(N+2)+j] = 0.25*(h*h*b[i*(N+2)+j] + x_old[(i-1)*(N+2)+j] + x_old[(i+1)*(N+2)+j] + x_old[i*(N+2)+j-1] + x_old[i*(N+2)+j+1]);
            }
        }
        // overwrite x_old with x
        for (int i=1; i <= N; i++) {
            for (int j=1; j <= N; j++) {
                x_old[i*(N+2)+j] = x[i*(N+2)+j];
            }
        }
    }
    free(x_old);
    free(b);
    return x;
}

__global__  void calculate_x(double* x_old, double* x, double* b, int N, double h) {
	// Find indices in grid
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	// Update x
	x[i*(N+2)+j] = 0.25*(h*h*b[i*(N+2)+j] + x_old[(i-1)*(N+2)+j] + x_old[(i+1)*(N+2)+j] + x_old[i*(N+2)+j-1] + x_old[i*(N+2)+j+1]);
}


double* jacobi_cuda(int N) {
  	// Allocate variables on host 
	double h = 1.0 / (double) N;
	double *b = initialize_b(N, h);
    double *x = initialize_x(N, h);
	
	// Allocate variables on device
	size_t arrsize = sizeof(double)*(N+2)*(N+2);
	double *d_b, *d_x_old, *d_x;
	hipMalloc((void **) &d_b, arrsize);
	hipMalloc((void **) &d_x_old, arrsize);
	hipMalloc((void **) &d_x, arrsize);
	
	// Copy data to device
	hipMemcpy(d_b, b, arrsize, hipMemcpyHostToDevice);
	hipMemcpy(d_x_old, x, arrsize, hipMemcpyHostToDevice);

	// We have 1024*1024 values to calculate in one iteration
	// 256 threads is a fairly safe number, thus the number of blocks required is 4096
	// We access these 2-dimensionally
	dim3 blocks(64,64,1);
	dim3 threads(16,16,1);
	
    for (int iteration=0; iteration < 2000; iteration++) {
		// Calculate new x values
        calculate_x<<<blocks,threads>>>(d_b, d_x_old, d_x, N, h);
        hipDeviceSynchronize();
		
		// Overwrite x_old with x once all kernels are finished
		double *tmp = d_x_old;
		d_x_old = d_x;
		d_x = tmp;
    }
	
	// Copy data to host
	hipMemcpy(x, d_x, arrsize, hipMemcpyDeviceToHost);
	
    free(b);
	hipFree(d_x_old);
	hipFree(d_b);
	hipFree(d_x);
    return x;
}

int main() {
    int N = 1024;
    
    auto start = chrono::high_resolution_clock::now();
    double* x = jacobi(N);
    auto end = chrono::high_resolution_clock::now();
    auto duration = chrono::duration_cast<chrono::milliseconds>(end-start);
    cout << "Done with Jacobi!" << endl;
    cout << "Time taken: " << duration.count() << "ms" << endl;

    start = chrono::high_resolution_clock::now();
    double* y = jacobi_cuda(N);
    end = chrono::high_resolution_clock::now();
    duration = chrono::duration_cast<chrono::milliseconds>(end-start);
    cout << "Done with Jacobi-CUDA!" << endl;
    cout << "Time taken: " << duration.count() << "ms" << endl;

    free(x);
    free(y);
}
/*
Ausgabe:
Done with Jacobi! Time taken: 9830ms
Done with Jacobi-CUDA! Time taken: 9306ms
*/